#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<algorithm>
#include<queue>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<omp.h>
#include<vector>
using namespace std;

#define BLOCK_SIZE 256
#define N_THREADS 8

int V,D,E,L,K,A,B,C,M,Q;
int* X;
int* edges;

int squared_l2_dist(int* x,int* y,int D){
	int sum2 = 0;
	for(int i = 0;i < D;++i)
		sum2 += (x[i] - y[i]) * (x[i] - y[i]);
	return sum2;
}

__global__ void squared_l2_dist_list(int* x,int* y, int* sum2, int D) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handling arbitrary vector size
    if (tid < D){
        sum2[tid] = (x[tid] - y[tid]) * (x[tid] - y[tid]);
    }
}


__global__ void squared_l2_dist_reduce_simple(int* g_idata, int* g_odata, int n) {

	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	// if (tid == 0) g_odata[blockIdx.x] = sdata[0];
	if (tid == 0)
        atomicAdd(g_odata, sdata[0]);

}


__global__ void squared_l2_dist_reduce_simple_combined(int* x, int* y, int* g_odata) {

	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	// sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];

	sdata[tid] = ((x[i] - y[i]) * (x[i] - y[i])) + ((x[i+blockDim.x] - y[i+blockDim.x]) * (x[i+blockDim.x] - y[i+blockDim.x]));
	
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	// if (tid == 0) g_odata[blockIdx.x] = sdata[0];
	if (tid == 0)
        atomicAdd(g_odata, sdata[0]);

}


__global__ void reduce_array(int* g_idata, int* g_odata) {

	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	// if (tid == 0) g_odata[blockIdx.x] = sdata[0];
	if (tid == 0)
        atomicAdd(g_odata, sdata[0]);

}

__global__ void get_array_to_sum(int* x, int* y, int* sum2, int D) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handling arbitrary vector size
    if (tid < D){
        sum2[tid] = (x[tid] - y[tid]) * (x[tid] - y[tid]);
    }
}


__global__ void get_each_distance(vector<int> d, int* y, vector<int> dist_list) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	int block_size = BLOCK_SIZE;
	int grid_size = ((D + block_size) / block_size);
	int *d_sum_list, *d_d;

	hipMalloc((void**)&d_sum_list, sizeof(int) * D);
	hipMalloc(&d_d, sizeof(int));

	get_array_to_sum<<<grid_size,block_size>>>(X + d[tid] * D, y, d_sum_list, D);
	reduce_array<<<grid_size,block_size>>>(d_sum_list, d_d);

	if (tid < d.size()){
		dist_list[tid] = *d_d;
	}

}


int nearest_id(int start_point,int max_hop,int* query_data){
	std::queue<std::pair<int,int>> q;
	q.push(std::make_pair(start_point,0));
	int min_d = std::numeric_limits<int>::max();
	int min_id = -1;

	vector<int> dist_id;
	vector<int> dist_list;
	int count = 0;

	while(!q.empty()){
		auto now = q.front();
		q.pop();
		int id = now.first;
		int hop = now.second;
		int d = 0;
		
		dist_id.push_back(id);
		// d = squared_l2_dist(X + id * D,query_data,D);

		// if((d < min_d) || (d == min_d && id < min_id)){
		// 	min_d = d;
		// 	min_id = id;
		// }
		if(hop + 1 <= max_hop){
			int degree = edges[id * (L + 1)];
			for(int i = 1;i <= degree;++i){
				int v = edges[id * (L + 1) + i];
				q.push(std::make_pair(v,hop + 1));
			}
		}

		count++;

	}

	// printf("%d ", dist_id.size());


	get_each_distance<<<1,2>>>(dist_id, query_data, dist_list);



	return min_id;
}

int main(int argc,char** argv){
	FILE* fin = fopen(argv[1],"r");
	FILE* fout = fopen(argv[2],"w");
	fscanf(fin,"%d%d%d%d%d%d%d%d%d%d",&V,&D,&E,&L,&K,&A,&B,&C,&M,&Q);
	X = new int[V * D];
	for(int i = 0;i < K;++i)
		fscanf(fin,"%d",&X[i]);
	for(int i = K;i < V * D;++i)
		X[i] = ((long long)A * X[i - 1] + (long long)B * X[i - 2] + C) % M;
	edges = new int[V * (L + 1)];
	for(int i = 0;i < V;++i){
		edges[i * (L + 1)] = 0;
	}
	for(int i = 0;i < E;++i){
		int u,v;
		fscanf(fin,"%d%d",&u,&v);
		int degree = edges[u * (L + 1)];
		edges[u * (L + 1) + degree + 1] = v;
		++edges[u * (L + 1)];
	}
	int* query_data = new int[D];


	// can we convert all vars to cuda in here?
	// vars: query_data


	int nid;



	for(int i = 0;i < Q;++i){
		int start_point,hop;
		fscanf(fin,"%d%d",&start_point,&hop);
		for(int i = 0;i < D;++i){
			fscanf(fin,"%d",&query_data[i]);
		}

		nid = nearest_id(start_point,hop,query_data);

		fprintf(fout,"%d\n", nid);
	}
	fclose(fin);
	fclose(fout);

	delete[] X;
	delete[] edges;
	delete[] query_data;

	return 0;
}

